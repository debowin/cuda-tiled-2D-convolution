#include "hip/hip_runtime.h"
/* Matrix multiplication: C = A * B.
 * Host code.
 */

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <sys/time.h>

#include "2Dconvolution.h"

// includes, kernels
__constant__ float Mc[KERNEL_SIZE * KERNEL_SIZE];

#include "2Dconvolution_kernel.cu"

////////////////////////////////////////////////////////////////////////////////
// declarations, forward

extern "C"
void computeGold(float*, const float*, const float*, unsigned int, unsigned int);

Matrix AllocateDeviceMatrix(const Matrix M);
Matrix AllocateMatrix(int height, int width, int init);
void CopyToDeviceMatrix(Matrix Mdevice, const Matrix Mhost);
void CopyFromDeviceMatrix(Matrix Mhost, const Matrix Mdevice);
int ReadFile(Matrix* M, char* file_name);
void WriteFile(Matrix M, char* file_name);
void FreeDeviceMatrix(Matrix* M);
void FreeMatrix(Matrix* M);
int ReadParamsFile(int* params, char* file_name, int num_params);
void ConvolutionOnDevice(const Matrix M, const Matrix N, Matrix P);
bool CompareMatrices(Matrix A, Matrix B);

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char** argv) {

	Matrix  M;
	Matrix  N;
	Matrix  P;
	
	srand(2012);
	
	if(argc != 5 && argc != 4) 
	{
		// Allocate and initialize the matrices
		M  = AllocateMatrix(KERNEL_SIZE, KERNEL_SIZE, 1);
		N  = AllocateMatrix((rand() % 1024) + 1, (rand() % 1024) + 1, 1);
		P  = AllocateMatrix(N.height, N.width, 0);
	}
	else
	{
		// Allocate and read in matrices from disk
		int* params = (int*) malloc(2*sizeof(int)); 
		unsigned int data_read = ReadParamsFile(params, argv[1], 2);
		if(data_read != 2){
			printf("Error reading parameter file\n");
			return 1;
		}

		M  = AllocateMatrix(KERNEL_SIZE, KERNEL_SIZE, 0);
		N  = AllocateMatrix(params[0], params[1], 0);		
		P  = AllocateMatrix(params[0], params[1], 0);
		free(params);
		(void)ReadFile(&M, argv[2]);
		(void)ReadFile(&N, argv[3]);
	}

	// M * N on the device
    ConvolutionOnDevice(M, N, P);
    
    // compute the matrix multiplication on the CPU for comparison
    Matrix reference = AllocateMatrix(P.height, P.width, 0);
    hipEvent_t start, stop;
    float elapsedTime = 0;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    computeGold(reference.elements, M.elements, N.elements, N.height, N.width);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("Time taken by CPU Gold %lf milliseconds.\n", elapsedTime);
        
    // in this case check if the result is equivalent to the expected soluion
    bool res = CompareMatrices(reference, P);
    printf("Test %s\n", (res) ? "PASSED" : "FAILED");
    
    if(argc == 5)
    {
		WriteFile(P, argv[4]);
	}
	else if(argc == 2)
	{
	    WriteFile(P, argv[1]);
	}   

	// Free matrices
    FreeMatrix(&M);
    FreeMatrix(&N);
    FreeMatrix(&P);
	return 0;
}


////////////////////////////////////////////////////////////////////////////////
//! Run a simple test for CUDA
////////////////////////////////////////////////////////////////////////////////
void ConvolutionOnDevice(const Matrix M, const Matrix N, Matrix P)
{
    hipEvent_t startBig, stopBig, start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventCreate(&startBig);
    hipEventCreate(&stopBig);
    hipEventRecord(startBig);
    // Load M and N to the device
    int size = M.width * M.height * sizeof(float);
    hipMemcpyToSymbol(HIP_SYMBOL(Mc), M.elements, size);
 
    Matrix Nd = AllocateDeviceMatrix(N);
    CopyToDeviceMatrix(Nd, N);

    // Allocate P on the device
    Matrix Pd = AllocateDeviceMatrix(P);

    // Setup the execution configuration
    dim3 blockSize, gridSize;
    blockSize.x = BLOCK_SIZE, blockSize.y = BLOCK_SIZE, blockSize.z = 1;
    gridSize.x = ceil((float)P.width/TILE_SIZE),
    gridSize.y = ceil((float)P.height/TILE_SIZE),
    gridSize.z = 1;

    // Launch the device computation threads!
    hipEventRecord(start);
    ConvolutionKernel<<<gridSize, blockSize>>>(Nd, Pd);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float kernelElapsedTime = 0;
    hipEventElapsedTime(&kernelElapsedTime, start, stop);
    printf("Time taken by GPU Kernel %lf milliseconds.\n", kernelElapsedTime);

    // Read P from the device
    CopyFromDeviceMatrix(P, Pd);

    // Free device matrices
    FreeDeviceMatrix(&Nd);
    FreeDeviceMatrix(&Pd);

    hipEventRecord(stopBig);
    hipEventSynchronize(stopBig);
    float elapsedTime = 0;
    hipEventElapsedTime(&elapsedTime, startBig, stopBig);
    printf("Overhead Time taken by GPU %lf milliseconds.\n", elapsedTime-kernelElapsedTime);
}

// Allocate a device matrix of same size as M.
Matrix AllocateDeviceMatrix(const Matrix M)
{
    Matrix Mdevice = M;
    int size = M.width * M.height * sizeof(float);
    hipMalloc((void**)&Mdevice.elements, size);
    return Mdevice;
}

// Allocate a device matrix of dimensions height*width
//	If init == 0, initialize to all zeroes.  
//	If init == 1, perform random initialization.
//  If init == 2, initialize matrix parameters, but do not allocate memory 
Matrix AllocateMatrix(int height, int width, int init)
{
    Matrix M;
    M.width = M.pitch = width;
    M.height = height;
    int size = M.width * M.height;
    M.elements = NULL;
    
    // don't allocate memory on option 2
    if(init == 2)
		return M;
		
	M.elements = (float*) malloc(size*sizeof(float));

	for(unsigned int i = 0; i < M.height * M.width; i++)
	{
		M.elements[i] = (init == 0) ? (0.0f) : (rand() / (float)RAND_MAX);
		if(rand() % 2)
			M.elements[i] = - M.elements[i];
	}
    return M;
}	

// Copy a host matrix to a device matrix.
void CopyToDeviceMatrix(Matrix Mdevice, const Matrix Mhost)
{
    int size = Mhost.width * Mhost.height * sizeof(float);
    Mdevice.height = Mhost.height;
    Mdevice.width = Mhost.width;
    Mdevice.pitch = Mhost.pitch;
    hipMemcpy(Mdevice.elements, Mhost.elements, size, 
					hipMemcpyHostToDevice);
}

// Copy a device matrix to a host matrix.
void CopyFromDeviceMatrix(Matrix Mhost, const Matrix Mdevice)
{
    int size = Mdevice.width * Mdevice.height * sizeof(float);
    hipMemcpy(Mhost.elements, Mdevice.elements, size, 
					hipMemcpyDeviceToHost);
}

// Free a device matrix.
void FreeDeviceMatrix(Matrix* M)
{
    hipFree(M->elements);
    M->elements = NULL;
}

// Free a host Matrix
void FreeMatrix(Matrix* M)
{
    free(M->elements);
    M->elements = NULL;
}

// Read a floating point matrix in from file
// Returns zero if the number of elements read is 
//  equals M.height * M.width, and 1 otherwise
int ReadFile(Matrix* M, char* file_name)
{
    unsigned int data_read = M->width * M->height;
    FILE* input = fopen(file_name, "r");
    for (unsigned i = 0; i < data_read; i++) 
        fscanf(input, "%f", &(M->elements[i]));
    return data_read;
}

// Read params of input matrices
int ReadParamsFile(int* params, char* file_name, int num_params)
{
    FILE* input = fopen(file_name, "r");
    for (unsigned i = 0; i < num_params; i++) 
        fscanf(input, "%d", &(params[i]));
    return num_params;
}

// Write a 16x16 floating point matrix to file
void WriteFile(Matrix M, char* file_name)
{
    unsigned int size = M.width * M.height;
    FILE* output = fopen(file_name, "w");
    for (unsigned i = 0; i < size; i++) {
        fprintf(output, "%f ", M.elements[i]);
    }
}

// returns true iff A and B have same elements in same order
bool CompareMatrices(Matrix A, Matrix B) {
    unsigned int size = A.width * A.height;

    if ( (A.width != B.width) || (A.height != B.height) )
        return false;

    for (unsigned i = 0; i < size; i++)
        if (abs(A.elements[i] - B.elements[i]) > 0.001f)
            return false;
    return true;
}

